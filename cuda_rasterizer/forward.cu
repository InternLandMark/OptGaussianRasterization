#include "hip/hip_runtime.h"
/*
* Copyright (C) 2023, Inria
* GRAPHDECO research group, https://team.inria.fr/graphdeco
* All rights reserved.
*
* This software is free for non-commercial, research and evaluation use 
* under the terms of the LICENSE.md file.
*
* For inquiries contact  george.drettakis@inria.fr
*/

#include "forward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include <fstream>
#include <algorithm>


// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, bool* clamped)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	// clamped[3 * idx + 0] = (result.x < 0);
	// clamped[3 * idx + 1] = (result.y < 0);
	// clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}

// Forward version of 2D covariance matrix computation
__device__ float3 computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, glm::mat4 viewmatrix)
{
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002). 
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	float3 t = transformPoint4x3(mean, (float*)&viewmatrix);

	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		((float*)&viewmatrix)[0], ((float*)&viewmatrix)[4], ((float*)&viewmatrix)[8],
		((float*)&viewmatrix)[1], ((float*)&viewmatrix)[5], ((float*)&viewmatrix)[9],
		((float*)&viewmatrix)[2], ((float*)&viewmatrix)[6], ((float*)&viewmatrix)[10]);

	glm::mat3 T = W * J;

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Apply low-pass filter: every Gaussian should be at least
	// one pixel wide/high. Discard 3rd row and column.
	cov[0][0] += 0.3f;
	cov[1][1] += 0.3f;
	return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
}

// Perform initial steps for each Gaussian prior to rasterization.
//计算每一个3d高斯球在每个平面上的投影，以及投影与平面tile的交集
template<int C>
__global__ void preprocessCUDA(int P, int D, int M,
	const float* orig_points,
	const float* opacities,
	const float* shs,
	glm::mat4 viewmatrix,
	glm::mat4 projmatrix,
	glm::vec3 cam_pos,
	const int W, int H,
	const float tan_fovx, float tan_fovy,
	const float focal_x, float focal_y,
	int* radii,
	float2* points_xy_image,
	float* depths,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	int* tiles_touched)
{
	//处理高斯球
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	int my_radii = 0;
	int my_tiles_touched = 0;

	do {
		// Perform near culling, quit if outside.
		float3 p_view;
		if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view))
			break;

		if (255.0f * opacities[idx] < 1.0f)
			break;
		
		//3d高斯球的中心点投影
		// Transform point by projecting
		float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
		float4 p_hom = transformPoint4x4(p_orig, (float*)&projmatrix);
		float p_w = 1.0f / (p_hom.w + 0.0000001f);
		float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

		//3d高斯投影在平面上的椭圆
		// Compute 2D screen-space covariance matrix
		float3 cov = computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3Ds + idx * 6, viewmatrix);

		// Invert covariance (EWA algorithm)
		float det = (cov.x * cov.z - cov.y * cov.y);
		// if (det == 0.0f)
		// 	return;
		float det_inv = 1.f / det;
		float3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };

		float power = __logf(255.0f * opacities[idx]);
		int width = (int)(1.414214f * __fsqrt_ru(cov.x * power) + 1.0f);
		int height = (int)(1.414214f * __fsqrt_ru(cov.z * power) + 1.0f);

		//圆覆盖了哪些tile，要用的值存储在中间结果中
		uint2 rect_min, rect_max;
		float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };
		getRect(point_image, width, height, rect_min, rect_max, grid);
		if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
			break;

		// If colors have been precomputed, use them, otherwise convert
		// spherical harmonics coefficients to RGB color.
		if(shs != nullptr)
		{
			glm::vec3 color = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, cam_pos, shs, nullptr);
			rgb[idx * C + 0] = color.x;
			rgb[idx * C + 1] = color.y;
			rgb[idx * C + 2] = color.z;
		}


		//计算深度值 depth id
		// Store some useful helper data for the next steps.
		depths[idx] = p_view.z;
		
		points_xy_image[idx] = point_image;
		// Inverse 2D covariance and opacity neatly pack into one float4
		conic_opacity[idx] = { conic.x, conic.y, conic.z, opacities[idx] };

		my_radii = width | height << 16;
		my_tiles_touched = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
	} while (false);

	radii[idx] = my_radii;
	tiles_touched[idx] = my_tiles_touched;
	// printf("proprecessend\n");
}

__device__ bool block_intersect_ellipse(int2 pix_min, int2 pix_max, float2 center, float4 con_o, bool check_x, bool check_y)
{
	float a, b, c, delta, dx, dy;
	int lambda1, lambda2;
	float w = 2.0f * __logf(256.0f * con_o.w);
	
	if (check_y)
	{
		dx = center.x - pix_min.x;
		a = con_o.z;
		b = -2.0f * con_o.y * dx;
		c = con_o.x * dx * dx - w;
		delta = b * b - 4.0f * a * c;
		if (delta >= 0.0f)
		{
			float sqrt_delta = __fsqrt_ru(delta);
			lambda1 = (-b + sqrt_delta) / (2.0f * a) + center.y;
			lambda2 = (-b - sqrt_delta) / (2.0f * a) + center.y;
			if (pix_min.y <= lambda1 && pix_max.y >= lambda2)
			{
				return true;
			}
		}
		
		dx = center.x - pix_max.x;
		a = con_o.z;
		b = -2.0f * con_o.y * dx;
		c = con_o.x * dx * dx - w;
		delta = b * b - 4.0f * a * c;
		if (delta >= 0.0f)
		{
			float sqrt_delta = __fsqrt_ru(delta);
			lambda1 = (-b + sqrt_delta) / (2.0f * a) + center.y;
			lambda2 = (-b - sqrt_delta) / (2.0f * a) + center.y;
			if (pix_min.y <= lambda1 && pix_max.y >= lambda2)
			{
				return true;
			}
		}			
	}

	if (check_x)
	{
		dy = center.y - pix_min.y;
		a = con_o.x;
		b = -2.0f * con_o.y * dy;
		c = con_o.z * dy * dy - w;
		delta = b * b - 4.0f * a * c;
		if (delta >= 0.0f)
		{
			float sqrt_delta = __fsqrt_ru(delta);
			lambda1 = (-b + sqrt_delta) / (2.0f * a) + center.x;
			lambda2 = (-b - sqrt_delta) / (2.0f * a) + center.x;
			if (pix_min.x <= lambda1 && pix_max.x >= lambda2)
			{
				return true;
			}
		}
		
		dy = center.y - pix_max.y;
		a = con_o.x;
		b = -2.0f * con_o.y * dy;
		c = con_o.z * dy * dy - w;
		delta = b * b - 4.0f * a * c;
		if (delta >= 0.0f)
		{
			float sqrt_delta = __fsqrt_ru(delta);
			lambda1 = (-b + sqrt_delta) / (2.0f * a) + center.x;
			lambda2 = (-b - sqrt_delta) / (2.0f * a) + center.x;
			if (pix_min.x <= lambda1 && pix_max.x >= lambda2)
			{
				return true;
			}
		}
	}
	return false;
}

__device__ bool block_contains_center(int2 pix_min, int2 pix_max, float2 center)
{
	return center.x >= pix_min.x && center.x <= pix_max.x && center.y >= pix_min.y && center.y <= pix_max.y;
}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching
// and rasterizing data.
template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float2* __restrict__ points_xy_image,
	const float* __restrict__ features,
	const float* __restrict__ depths,
	const float4* __restrict__ conic_opacity,
	float* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	const float* __restrict__ bg_color,
	float* __restrict__ out_color,
	float* __restrict__ out_depth)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	int2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	int2 pix_max = { min(pix_min.x + BLOCK_X - 1, W), min(pix_min.y + BLOCK_Y - 1, H) };
	int2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y };

	// Check if this thread is associated with a valid pixel or outside.
	bool inside;

	inside = pix.x < W && pix.y < H;
	
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];
	__shared__ int is_valid[BLOCK_SIZE];
	
	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS] = { 0 };
	float D = { 0 };

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			float2 my_points_xy_image = points_xy_image[coll_id];
			collected_xy[block.thread_rank()] = my_points_xy_image;
			float4 my_conic_opacity = conic_opacity[coll_id];
			collected_conic_opacity[block.thread_rank()] = my_conic_opacity;
			int valid_mask = 0;
			for (int k = 0; k < 16; k += 2)
			{
				int2 pix_min_warp = { pix_min.x , pix_min.y + k};
				int2 pix_max_warp = { pix_max.x , pix_min.y + k + 1};
				bool valid0 = block_contains_center(pix_min_warp, pix_max_warp, my_points_xy_image) || 
					block_intersect_ellipse(pix_min_warp, pix_max_warp, my_points_xy_image, my_conic_opacity, true, false);
				valid_mask |= valid0 ? (3 << k) : 0;
			}
			is_valid[block.thread_rank()] = valid_mask;
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;
			if (!(is_valid[j] & (1 << block.thread_index().y)))
				continue;

			// Resample using conic matrix (cf. "Surface
			// Splatting" by Zwicker et al., 2001)
			float2 xy = collected_xy[j];
			float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o = collected_conic_opacity[j];
			float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;
			
			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix).
			float alpha = min(0.99f, con_o.w * exp(power));
			if (alpha < 1.0f / 255.0f)
				continue;

			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}

			// Eq. (3) from 3D Gaussian splatting paper.
			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += features[collected_id[j] * CHANNELS + ch] * (alpha * T);
			D += depths[collected_id[j]] * (alpha * T);
			
			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
		out_depth[pix_id] = D;
	}
}

void FORWARD::render(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float2* means2D,
	const float* colors,
	const float* depths,
	const float4* conic_opacity,
	float* final_T,
	uint32_t* n_contrib,
	const float* bg_color,
	float* out_color,
	float* out_depth)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> > (
		ranges,
		point_list,
		W, H,
		means2D,
		colors,
		depths,
		conic_opacity,
		final_T,
		n_contrib,
		bg_color,
		out_color,
		out_depth);
}

// void FORWARD::preprocess(int P, int D, int M,
// 	const float* means3D,
// 	const glm::vec3* scales,
// 	const float scale_modifier,
// 	const glm::vec4* rotations,
// 	const float* opacities,
// 	const float* shs,
// 	bool* clamped,
// 	const float* cov3D_precomp,
// 	const float* colors_precomp,
// 	const float* viewmatrix,
// 	const float* projmatrix,
// 	const glm::vec3* cam_pos,
// 	const int W, int H,
// 	const float focal_x, float focal_y,
// 	const float tan_fovx, float tan_fovy,
// 	int* radii,
// 	float2* means2D,
// 	float* depths,
// 	float* cov3Ds,
// 	float* rgb,
// 	float4* conic_opacity,
// 	const dim3 grid,
// 	uint32_t* tiles_touched,
// 	bool prefiltered)
// {
// 	glm::mat4 viewmatrix_host;
// 	glm::mat4 projmatrix_host;
// 	glm::vec3 cam_pos_host;
// 	hipMemcpy(&viewmatrix_host, viewmatrix, sizeof viewmatrix_host, hipMemcpyDeviceToHost);
// 	hipMemcpy(&projmatrix_host, projmatrix, sizeof projmatrix_host, hipMemcpyDeviceToHost);
// 	hipMemcpy(&cam_pos_host, cam_pos, sizeof cam_pos_host, hipMemcpyDeviceToHost);

// 	preprocessCUDA<NUM_CHANNELS> << <(P + 255) / 256, 256 >> > (
// 		P, D, M,
// 		means3D,
// 		opacities,
// 		shs,
// 		viewmatrix_host, 
// 		projmatrix_host,
// 		cam_pos_host,
// 		W, H,
// 		tan_fovx, tan_fovy,
// 		focal_x, focal_y,
// 		radii,
// 		means2D,
// 		depths,
// 		cov3Ds,
// 		rgb,
// 		conic_opacity,
// 		grid,
// 		tiles_touched
// 		);
// }
void FORWARD::preprocess(
	int P, int D, int M,
	const float* orig_points,
	const float* opacities,
	const float* shs,
	glm::mat4 viewmatrix,
	glm::mat4 projmatrix,
	glm::vec3 cam_pos,
	const int W, int H,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	int* radii,
	float2* means2D,
	float* depths,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	int* tiles_touched)
{
	preprocessCUDA<NUM_CHANNELS> << <(P + 255) / 256, 256 >> > (
		P, D, M,
		orig_points,
		opacities,
		shs,
		viewmatrix, 
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		means2D,
		depths,
		cov3Ds,
		rgb,
		conic_opacity,
		grid,
		tiles_touched);
}
// Forward version of 2D covariance matrix computation
__device__ float3 origin_computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, const float* viewmatrix)
{
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002). 
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	float3 t = transformPoint4x3(mean, viewmatrix);

	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

	glm::mat3 T = W * J;

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Apply low-pass filter: every Gaussian should be at least
	// one pixel wide/high. Discard 3rd row and column.
	cov[0][0] += 0.3f;
	cov[1][1] += 0.3f;
	return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
}


// Forward method for converting scale and rotation properties of each
// Gaussian to a 3D covariance matrix in world space. Also takes care
// of quaternion normalization.
__device__ void origin_computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float* cov3D)
{
	// Create scaling matrix
	glm::mat3 S = glm::mat3(1.0f);
	S[0][0] = mod * scale.x;
	S[1][1] = mod * scale.y;
	S[2][2] = mod * scale.z;

	// Normalize quaternion to get valid rotation
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	// Compute rotation matrix from quaternion
	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 M = S * R;

	// Compute 3D world covariance matrix Sigma
	glm::mat3 Sigma = glm::transpose(M) * M;

	// Covariance is symmetric, only store upper right
	cov3D[0] = Sigma[0][0];
	cov3D[1] = Sigma[0][1];
	cov3D[2] = Sigma[0][2];
	cov3D[3] = Sigma[1][1];
	cov3D[4] = Sigma[1][2];
	cov3D[5] = Sigma[2][2];
}

// Perform initial steps for each Gaussian prior to rasterization.
template<int C>
__global__ void filter_preprocessCUDA(int P, int M,
	const float* orig_points,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const int W, int H,
	const float tan_fovx, float tan_fovy,
	const float focal_x, float focal_y,
	int* radii,
	float* cov3Ds,
	const dim3 grid,
	bool prefiltered)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;

	// Perform near culling, quit if outside.
	float3 p_view;
	if (!origin_in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;

	// Transform point by projecting
	float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

	// If 3D covariance matrix is precomputed, use it, otherwise compute
	// from scaling and rotation parameters. 
	const float* cov3D;
	if (cov3D_precomp != nullptr)
	{
		cov3D = cov3D_precomp + idx * 6;
	}
	else
	{
		origin_computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		cov3D = cov3Ds + idx * 6;
	}

	// Compute 2D screen-space covariance matrix
	float3 cov = origin_computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);

	// Invert covariance (EWA algorithm)
	float det = (cov.x * cov.z - cov.y * cov.y);
	if (det == 0.0f)
		return;


	// Compute extent in screen space (by finding eigenvalues of
	// 2D covariance matrix). Use extent to compute a bounding rectangle
	// of screen-space tiles that this Gaussian overlaps with. Quit if
	// rectangle covers 0 tiles. 
	float mid = 0.5f * (cov.x + cov.z);
	float lambda1 = mid + sqrt(max(0.1f, mid * mid - det));
	float lambda2 = mid - sqrt(max(0.1f, mid * mid - det));
	float my_radius = ceil(3.f * sqrt(max(lambda1, lambda2)));
	float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };
	uint2 rect_min, rect_max;
	origin_getRect(point_image, my_radius, rect_min, rect_max, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;


	radii[idx] = my_radius;
}


void FORWARD::filter_preprocess(int P, int M,
	const float* means3D,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const int W, int H,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	int* radii,
	float* cov3Ds,
	const dim3 grid,
	bool prefiltered)
{

	filter_preprocessCUDA<NUM_CHANNELS> << <(P + 255) / 256, 256 >> > (
		P, M,
		means3D,
		scales,
		scale_modifier,
		rotations,
		cov3D_precomp,
		viewmatrix, 
		projmatrix,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		cov3Ds,
		grid,
		prefiltered
		);
}
